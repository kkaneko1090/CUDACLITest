#include "hip/hip_runtime.h"
#pragma once
#include "Kernel.cuh"

/// <summary>
/// �x�N�g���a���v�Z����J�[�l���֐�
/// </summary>
/// <param name="vec_0">�x�N�g��0</param>
/// <param name="vec_1">�x�N�g��1</param>
/// <param name="result">�v�Z���ʂ̃x�N�g��</param>
/// <param name="length">�x�N�g���̒���</param>
/// <returns></returns>
__global__ void CudaAddKernel(float* vec_0, float* vec_1, float* result, int* length) {
	//�C���f�b�N�X
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	//�C���f�b�N�X���͈͓��̂Ƃ�
	if (index < *length) {
		//�x�N�g���̗v�f�ǂ����𑫂����킹��
		result[index] = vec_0[index] + vec_1[index];
	}
}